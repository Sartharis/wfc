#include "hip/hip_runtime.h"
#include <algorithm>
#include <array>
#include <cmath>
#include <limits>
#include <memory>
#include <numeric>
#include <random>
#include <unordered_map>
#include <unordered_set>
#include <vector>
#include <chrono>

#include <configuru.hpp>
#include <emilib/irange.hpp>
#include <emilib/strprintf.hpp>
#include <loguru.hpp>
#include <stb_image.h>
#include <stb_image_write.h>

#define JO_GIF_HEADER_FILE_ONLY
#include <jo_gif.cpp>

#include "arrays.hpp"

#include <hip/hip_runtime.h>

// ----------------------------------------------------------------------------

bool TileModel::propagate(Output* output) const
{
	bool did_change = false;

	for (int x2 = 0; x2 < _width; ++x2) {
		for (int y2 = 0; y2 < _height; ++y2) {
			for (int d = 0; d < 4; ++d) {
				int x1 = x2, y1 = y2;
				
				// Grab coordinates for given neighbor (periodic assumes pattern repeats over border)
				if (d == 0) {
					if (x2 == 0) {
						if (!_periodic_out) { continue; }
						x1 = _width - 1;
					} else {
						x1 = x2 - 1;
					}
				} else if (d == 1) {
					if (y2 == _height - 1) {
						if (!_periodic_out) { continue; }
						y1 = 0;
					} else {
						y1 = y2 + 1;
					}
				} else if (d == 2) {
					if (x2 == _width - 1) {
						if (!_periodic_out) { continue; }
						x1 = 0;
					} else {
						x1 = x2 + 1;
					}
				} else {
					if (y2 == 0) {
						if (!_periodic_out) { continue; }
						y1 = _height - 1;
					} else {
						y1 = y2 - 1;
					}
				}

				// If neighbor tile didn't change, skip it
				if (!output->_changes.get(x1, y1)) { continue; }

				for (int t2 = 0; t2 < _num_patterns; ++t2) {
					// if a pattern in our cell is still possible...
					if (output->_wave.get(x2, y2, t2)) {
						
						// ... check if the pattern is still valid for some possible pattern in neighbor ...
						bool b = false;
						for (int t1 = 0; t1 < _num_patterns && !b; ++t1) {
							if (output->_wave.get(x1, y1, t1)) {
								b = _propagator.get(d, t1, t2);
							}
						}

						// ... if not, mark that pattern as impossible
						if (!b) {
							output->_wave.set(x2, y2, t2, false);
							output->_changes.set(x2, y2, true);
							did_change = true;
						}
					}
				}
			}
		}
	}

	return did_change;
}

Image TileModel::image(const Output& output) const
{
	Image result(_width * _tile_size, _height * _tile_size, {});

	for (int x = 0; x < _width; ++x) {
		for (int y = 0; y < _height; ++y) {
			double sum = 0;
			for (const auto t : irange(_num_patterns)) {
				if (output._wave.get(x, y, t)) {
					sum += _pattern_weight[t];
				}
			}

			for (int yt = 0; yt < _tile_size; ++yt) {
				for (int xt = 0; xt < _tile_size; ++xt) {
					if (sum == 0) {
						result.set(x * _tile_size + xt, y * _tile_size + yt, RGBA{0, 0, 0, 255});
					} else {
						double r = 0, g = 0, b = 0, a = 0;
						for (int t = 0; t < _num_patterns; ++t) {
							if (output._wave.get(x, y, t)) {
								RGBA c = _tiles[t][xt + yt * _tile_size];
								r += (double)c.r * _pattern_weight[t] / sum;
								g += (double)c.g * _pattern_weight[t] / sum;
								b += (double)c.b * _pattern_weight[t] / sum;
								a += (double)c.a * _pattern_weight[t] / sum;
							}
						}

						result.set(x * _tile_size + xt, y * _tile_size + yt,
						           RGBA{(uint8_t)r, (uint8_t)g, (uint8_t)b, (uint8_t)a});
					}
				}
			}
		}
	}

	return result;
}

// ----------------------------------------------------------------------------


Result find_lowest_entropy(const Model& model, const Output& output, RandomDouble& random_double,
                           int* argminx, int* argminy)
{
	// We actually calculate exp(entropy), i.e. the sum of the weights of the possible patterns

	double min = std::numeric_limits<double>::infinity();

	// Iterate over every cell
	for (int x = 0; x < model._width; ++x) {
		for (int y = 0; y < model._height; ++y) {
			if (model.on_boundary(x, y)) { continue; }

			size_t num_superimposed = 0;
			double entropy = 0;

			//Get total entropy from the number of weighted possibilities
			for (int t = 0; t < model._num_patterns; ++t) {
				if (output._wave.get(x, y, t)) {
					num_superimposed += 1;
					entropy += model._pattern_weight[t];
				}
			}

			// If entropy is 0 / no options available, it means we reached an impossible state
			if (entropy == 0 || num_superimposed == 0) {
				return Result::kFail;
			}

			if (num_superimposed == 1) {
				continue; // Already frozen
			}

			// Add a tie-breaking bias:
			const double noise = 0.5 * random_double();
			entropy += noise;

			if (entropy < min) {
				min = entropy;
				*argminx = x;
				*argminy = y;
			}
		}
	}

	if (min == std::numeric_limits<double>::infinity()) {
		return Result::kSuccess;
	} else {
		return Result::kUnfinished;
	}
}

Result observe(const Model& model, Output* output, RandomDouble& random_double)
{
	int argminx, argminy;

	// Try to find the cell with lowest possible choices for pattern
	const auto result = find_lowest_entropy(model, *output, random_double, &argminx, &argminy);
	if (result != Result::kUnfinished) { return result; }

	// Create a probability distribution from all the possible patterns in the given cell
	std::vector<double> distribution(model._num_patterns);
	for (int t = 0; t < model._num_patterns; ++t) {
		distribution[t] = output->_wave.get(argminx, argminy, t) ? model._pattern_weight[t] : 0;
	}

	// Grab a random weighted pattern and choose a single pattern to use (i.e. collapse the cell)
	size_t r = spin_the_bottle(std::move(distribution), random_double());
	for (int t = 0; t < model._num_patterns; ++t) {
		output->_wave.set(argminx, argminy, t, t == r);
	}

	// Mark the collapsed cell as changed for propagation purposes
	output->_changes.set(argminx, argminy, true);

	return Result::kUnfinished;
}





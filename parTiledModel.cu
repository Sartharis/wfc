#include "hip/hip_runtime.h"
#include <algorithm>
#include <array>
#include <cmath>
#include <limits>
#include <memory>
#include <numeric>
#include <random>
#include <unordered_map>
#include <unordered_set>
#include <vector>
#include <chrono>

// #include <configuru.hpp>
// #include <emilib/irange.hpp>
// #include <emilib/strprintf.hpp>
// #include <loguru.hpp>
// #include <stb_image.h>
// #include <stb_image_write.h>

#define JO_GIF_HEADER_FILE_ONLY
// #include <jo_gif.cpp>

#include "arrays.hpp"

// ----------------------------------------------------------------------------

__global__ void pixelDidChange(bool* didChange, const int imgWidth, 
        const int imgHeight) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int x2 = index % imgWidth;
    int y2 = index / imgWidth;
    
    if (x2 >= imgWidth || y2 >= imgHeight) { return; }

    for (int dir=0; dir<4; dir++) {
        // Grab coordinates for neighbor in direction `dir`
        int x1 = x2, y1 = y2;
				
		// Grab coordinates for given neighbor (periodic assumes pattern repeats over border)
		if (dir == 0) {
            if (x2 == 0) {
                if (!_periodic_out) { continue; }
                x1 = imgWidth - 1;
            } else {
                x1 = x2 - 1;
            }
        } else if (dir == 1) {
            if (y2 == imgHeight - 1) {
                if (!_periodic_out) { continue; }
                y1 = 0;
            } else {
                y1 = y2 + 1;
            }
        } else if (dir == 2) {
            if (x2 == imgWidth - 1) {
                if (!_periodic_out) { continue; }
                x1 = 0;
            } else {
                x1 = x2 + 1;
            }
        } else {
            if (y2 == 0) {
                if (!_periodic_out) { continue; }
                y1 = imgHeight - 1;
            } else {
                y1 = y2 - 1;
            }
        }

        // If neighbor tile didn't change, skip it
        if (!output->_changes.get(x1, y1)) { continue; }

        for (int t2 = 0; t2 < _num_patterns; ++t2) {
            // if a pattern in our cell is still possible...
            if (output->_wave.get(x2, y2, t2)) {
                
                // ... check if the pattern is still valid for some possible pattern in neighbor ...
                bool b = false;
                for (int t1 = 0; t1 < _num_patterns && !b; ++t1) {
                    if (output->_wave.get(x1, y1, t1)) {
                        b = _propagator.get(d, t1, t2);
                    }
                }

                // ... if not, mark that pattern as impossible
                if (!b) {
                    output->_wave.set(x2, y2, t2, false);
                    output->_changes.set(x2, y2, true);
                    didChange[index] = true;
                }
            }
        }
    }
}


bool* TileModel::propagate(Output* output) const
{
    int gridSize = ;        // number of blocks in grid
	int blockSize = 256;    // number of threads per block
    bool* didChange = (bool*)malloc(sizeof(bool) * _width * _height);

    pixelDidChange<<<gridSize, blockSize>>>(didChangeFlags, _width, _height);
	return didChangeFlags;
}

Image TileModel::image(const Output& output) const
{
	Image result(_width * _tile_size, _height * _tile_size, {});

	for (int x = 0; x < _width; ++x) {
		for (int y = 0; y < _height; ++y) {
			double sum = 0;
			for (const auto t : irange(_num_patterns)) {
				if (output._wave.get(x, y, t)) {
					sum += _pattern_weight[t];
				}
			}

			for (int yt = 0; yt < _tile_size; ++yt) {
				for (int xt = 0; xt < _tile_size; ++xt) {
					if (sum == 0) {
						result.set(x * _tile_size + xt, y * _tile_size + yt, RGBA{0, 0, 0, 255});
					} else {
						double r = 0, g = 0, b = 0, a = 0;
						for (int t = 0; t < _num_patterns; ++t) {
							if (output._wave.get(x, y, t)) {
								RGBA c = _tiles[t][xt + yt * _tile_size];
								r += (double)c.r * _pattern_weight[t] / sum;
								g += (double)c.g * _pattern_weight[t] / sum;
								b += (double)c.b * _pattern_weight[t] / sum;
								a += (double)c.a * _pattern_weight[t] / sum;
							}
						}

						result.set(x * _tile_size + xt, y * _tile_size + yt,
						           RGBA{(uint8_t)r, (uint8_t)g, (uint8_t)b, (uint8_t)a});
					}
				}
			}
		}
	}

	return result;
}




#include "hip/hip_runtime.h"
#include <algorithm>
#include <array>
#include <cmath>
#include <limits>
#include <memory>
#include <numeric>
#include <random>
#include <unordered_map>
#include <unordered_set>
#include <vector>
#include <chrono>

#include <configuru.hpp>
#include <emilib/irange.hpp>
#include <emilib/strprintf.hpp>
#include <loguru.hpp>
#include <stb_image.h>
#include <stb_image_write.h>

#define JO_GIF_HEADER_FILE_ONLY
#include <jo_gif.cpp>

#include "arrays.hpp"

#include <hip/hip_runtime.h>

// ----------------------------------------------------------------------------

Result find_lowest_entropy(const Model& model, const Output& output, RandomDouble& random_double,
                           int* argminx, int* argminy)
{
	// We actually calculate exp(entropy), i.e. the sum of the weights of the possible patterns

	double min = std::numeric_limits<double>::infinity();

	// Iterate over every cell
	for (int x = 0; x < model._width; ++x) {
		for (int y = 0; y < model._height; ++y) {
			if (model.on_boundary(x, y)) { continue; }

			size_t num_superimposed = 0;
			double entropy = 0;

			//Get total entropy from the number of weighted possibilities
			for (int t = 0; t < model._num_patterns; ++t) {
				if (output._wave.get(x, y, t)) {
					num_superimposed += 1;
					entropy += model._pattern_weight[t];
				}
			}

			// If entropy is 0 / no options available, it means we reached an impossible state
			if (entropy == 0 || num_superimposed == 0) {
				return Result::kFail;
			}

			if (num_superimposed == 1) {
				continue; // Already frozen
			}

			// Add a tie-breaking bias:
			const double noise = 0.5 * random_double();
			entropy += noise;

			if (entropy < min) {
				min = entropy;
				*argminx = x;
				*argminy = y;
			}
		}
	}

	if (min == std::numeric_limits<double>::infinity()) {
		return Result::kSuccess;
	} else {
		return Result::kUnfinished;
	}
}

Result observe(const Model& model, Output* output, RandomDouble& random_double)
{
	int argminx, argminy;

	// Try to find the cell with lowest possible choices for pattern
	const auto result = find_lowest_entropy(model, *output, random_double, &argminx, &argminy);
	if (result != Result::kUnfinished) { return result; }

	// Create a probability distribution from all the possible patterns in the given cell
	std::vector<double> distribution(model._num_patterns);
	for (int t = 0; t < model._num_patterns; ++t) {
		distribution[t] = output->_wave.get(argminx, argminy, t) ? model._pattern_weight[t] : 0;
	}

	// Grab a random weighted pattern and choose a single pattern to use (i.e. collapse the cell)
	size_t r = spin_the_bottle(std::move(distribution), random_double());
	for (int t = 0; t < model._num_patterns; ++t) {
		output->_wave.set(argminx, argminy, t, t == r);
	}

	// Mark the collapsed cell as changed for propagation purposes
	output->_changes.set(argminx, argminy, true);

	return Result::kUnfinished;
}




